#include <complex>
#include <cassert>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <cstdio> // 使用 C++ 风格的头文件

using namespace std;
static constexpr float PI=3.14159265358979323846;


struct DDCResources2 //secondary ddc
{
    int N;  // 每次追加的数据长度
    int NDEC;
    int K;
    hipFloatComplex* d_indata;
    hipFloatComplex *d_outdata;
    hipFloatComplex *gpu_buffer;
    float *d_fir_coeffs;
};



// 复数乘法
static __device__ hipFloatComplex complex_mult(float a, float b, float c, float d)
{
    return make_hipFloatComplex(a * c - b * d, a * d + b * c);
}

__global__ void mix2(hipFloatComplex *indata, hipFloatComplex *gpu_buffer, int offset, int N, int lo_ch)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
    {
        float phase=-(float)i*(float)lo_ch/(float)N*2.0*PI;
        float lo_cos=cos(phase);
        float lo_sin=sin(phase);
        gpu_buffer[offset + i] = complex_mult(indata[i].x, indata[i].y, lo_cos, lo_sin);
    }
}

__global__ void fir_filter2(hipFloatComplex *gpu_buffer, hipFloatComplex *outdata, const float *fir_coeffs, int NDEC, int K, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int output_index = i;
    int input_index = i * NDEC;

    if (output_index < N / NDEC)
    {
        hipFloatComplex sum = make_hipFloatComplex(0.0f, 0.0f);
        for (int j = 0; j < K * NDEC; j++)
        {
            sum = hipCaddf(sum, hipCmulf(make_hipFloatComplex(fir_coeffs[j], 0.0f), gpu_buffer[input_index + j]));
        }
        outdata[output_index] = sum;
    }
}

// 初始化 DDC 资源
extern "C" void init_ddc_resources2(DDCResources2 *res,int N, int NDEC, int K, const float *fir_coeffs)
{
    res->NDEC = NDEC;
    res->K = K;
    res->N=N;
    int buffer_size =  N + NDEC * (K - 1);
    int fir_size = NDEC * K;

    hipError_t err = hipMalloc((void **)&res->d_indata, N * sizeof(hipFloatComplex));
    assert(err == hipSuccess);
    err = hipMalloc((void **)&res->d_outdata, (N / NDEC) * sizeof(hipFloatComplex));
    assert(err == hipSuccess);
    err = hipMalloc((void **)&res->gpu_buffer, buffer_size * sizeof(hipFloatComplex));
    assert(err == hipSuccess);
    err = hipMalloc((void **)&res->d_fir_coeffs, fir_size * sizeof(float));
    assert(err == hipSuccess);

    err = hipMemcpy(res->d_fir_coeffs, fir_coeffs, fir_size * sizeof(float), hipMemcpyHostToDevice);
    assert(err == hipSuccess);
}

// 释放资源
extern "C" void free_ddc_resources2(DDCResources2 *res)
{
    hipFree(res->d_indata);
    hipFree(res->d_outdata);
    hipFree(res->gpu_buffer);
    hipFree(res->d_fir_coeffs);
}

// DDC 处理
extern "C" int ddc2(const hipFloatComplex *indata, int lo_ch, DDCResources2 *res)
{

        //int buffer_size = total_size + res->NDEC * (res->K - 1);
        int offset = res->NDEC * (res->K - 1);

        hipMemcpy(res->d_indata, indata, res->N * sizeof(hipFloatComplex), hipMemcpyDeviceToDevice);
        mix2<<<(res->N + 255) / 256, 256>>>(res->d_indata, res->gpu_buffer, offset, res->N, lo_ch);
        hipError_t err = hipGetLastError();
        if (err != hipSuccess)
            return -1;

        hipDeviceSynchronize();
        err = hipGetLastError();
        if (err != hipSuccess)
            return -1;

        fir_filter2<<<(res->N / res->NDEC + 255) / 256, 256>>>(res->gpu_buffer, res->d_outdata, res->d_fir_coeffs, res->NDEC, res->K, res->N);
        err = hipGetLastError();
        if (err != hipSuccess)
            return -1;
        hipDeviceSynchronize();
        err = hipGetLastError();
        if (err != hipSuccess)
            return -1;
        return 1;
}

extern "C" void fetch_output2(std::complex<float> *outdata, DDCResources2* res){
    hipMemcpy(outdata, res->d_outdata, (res->N / res->NDEC) * sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
}


extern "C" int calc_output_size2(const DDCResources2* res){
    return (res->N)/(res->NDEC);
}
